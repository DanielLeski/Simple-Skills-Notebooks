#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    //handling normal sizes for vectors
    //for(int i = 0; i < n; i++){
    //    out[i] = a[i] + b[i];
    //}

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        tid[out] = a[tid] + b[tid]
    }

}

int main() {
    //allocation of current host 
    float *a, *b, *out;
    //allocation memory for the device
    float *d_a, *d_b, *d_out;

    //Allocate this space in memory for each variable
    a = (float*)malloc(sizeof(float)*N);
    b = (float*)malloc(sizeof(float)*N);
    out = (float*)malloc(sizeof(float)*N);

    for(int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }
    
    /// Allocate memory on the device 
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc(void(**)&d_out, sizeof(float) * N);

    //Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);


    // Executing kernel 
    //max M size is 255
    //max T size is 256
    int blocksize = 256
    int grid_size = ((N + blocksize) / block_size);
    vector_add<<<grid_size,block_size>>>(d_out, d_a, d_b, N);
    
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < N; i++) {
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
//
    printf("PASSED\n");

    //Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    //Deallocate host memory
    free(a);
    free(b);
    free(out);

    return 0;
}